#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include "common.h"

extern const int DEFAULT_SIGNAL_LENGTH;
extern const int DEFAULT_FFT_TRIALS;
extern const int DEFAULT_META_TRIALS;

const int BATCH_SIZE = 1;

int main(int argc, char **argv) {
    int fft_trials = DEFAULT_FFT_TRIALS;
    int meta_trials = DEFAULT_META_TRIALS;
    if (argc >= 2) {
        char *arg_fft_trials = argv[1];
        char *invalid_chars;
        fft_trials = strtol(arg_fft_trials, &invalid_chars, 10);
        if (*invalid_chars != '\0') {
            fprintf(stderr, "[ERROR] FFT trials number must be integer (but given: '%s')\n", arg_fft_trials);
            return 1;
        }

        if (argc >= 3) {
            char *arg_meta_trials = argv[2];
            char *invalid_chars;
            meta_trials = strtol(arg_meta_trials, &invalid_chars, 10);
            if (*invalid_chars != '\0') {
                fprintf(stderr, "[ERROR] Meta trials number must be integer (but given: '%s')\n", arg_meta_trials);
                return 1;
            }
        }
    }
    printf("[INFO] META trials: %d\n", meta_trials);
    printf("[INFO] FFT trials: %d\n", fft_trials);

    long signal_length = DEFAULT_SIGNAL_LENGTH;
    char *env_signal_length = getenv("SIGNAL_LENGTH");
    if (env_signal_length != NULL) {
        char *invalid_chars;
        signal_length = strtol(env_signal_length, &invalid_chars, 10);
        if (*invalid_chars != '\0') {
            fprintf(stderr, "[ERROR] Environment variable of 'SIGNAL_LENGTH' must be integer (but given: '%s')\n", env_signal_length);
            return 1;
        }
    }
    printf("[INFO] Signal Length: %ld\n", signal_length);

    hipfftComplex *h_original_signal;
    hipHostMalloc((void **) &h_original_signal, sizeof(hipfftComplex) * signal_length);

    hipfftComplex *d_original_signal, *d_applied_fft_signal;
    hipMalloc((void **) &d_original_signal, sizeof(hipfftComplex) * signal_length);
    hipMalloc((void **) &d_applied_fft_signal, sizeof(hipfftComplex) * signal_length);

    /*
     * generate random signal as original signal
     */
    srand(time(NULL)); // initialize random seed
    for (int i = 0; i < signal_length; i++) {
        h_original_signal[i].x = (float)rand() / RAND_MAX;
        h_original_signal[i].y = 0.0;
    }
    hipMemcpy(d_original_signal, h_original_signal, sizeof(hipfftComplex) * signal_length, hipMemcpyHostToDevice);

    hipfftHandle fft_plan;
    hipfftPlan1d(&fft_plan, signal_length, HIPFFT_C2C, BATCH_SIZE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float sum_of_elapsed_times = 0.0;

    printf("[INFO] Run benchmark...\n");
    for (int i = 0; i < meta_trials; i++) {
        hipEventRecord(start, 0);

        for (int j = 0; j < fft_trials; j++) {
            hipfftExecC2C(fft_plan, d_original_signal, d_applied_fft_signal, HIPFFT_FORWARD);
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float elapsed_time_ms;
        hipEventElapsedTime(&elapsed_time_ms, start, stop);

        float elapsed_time_sec = elapsed_time_ms / 1000.0;
        sum_of_elapsed_times += elapsed_time_sec;
        printf("%f sec\n", elapsed_time_sec);
    }
    printf("[INFO] Finished!\n");
    printf("[INFO] Average: %lf sec\n", sum_of_elapsed_times / meta_trials);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}
